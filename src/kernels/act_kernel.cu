#include "hip/hip_runtime.h"
#include <iostream>
#include "src/kernels/act_kernel.h"
#include "src/utils/cuda_debug_utils.cuh"
#include "src/utils/macro.h"
// fp32 silu version
template<typename T>
__device__ __forceinline__ T silu(const T& in) {
  // x * sigmoid(x)
  return (T) (((float) in) / (1.0f + expf((float) -in)));
}
// fp16 silu version
template<>
__device__ __forceinline__ half2 silu<half2>(const half2& in) {
  return make_half2(__float2half(silu<float>((float)(in.x))), __float2half(silu<float>((float)(in.y))));
}

// 代码逻辑：第一个intermediate 去做silu，其结果与第二个intermediate 做点乘mul
template<typename T>
__global__ void silu_and_mul_kernel(
  T* out,               // shape: [bs, intermedia size]
  const T* input,       // shape: [bs, 2, intermedia size]
  const int intermedia_size) {
  const int batch_idx = blockIdx.x;
  // 循环处理intermedia_size，当线程不够时，使得也能处理完
  for (int idx = threadIdx.x; idx < intermedia_size; idx += blockDim.x) { 
    // 第一个和第二个intermediate处于同一buffer: input
    // 根据shape索引第一个intermediate
    const T x = input[batch_idx * 2 * intermedia_size + idx];
    // 根据shape索引第二个intermediate
    const T y = input[batch_idx * 2 * intermedia_size + intermedia_size + idx];
    // 索引到了后做计算，把计算结果写回output
    out[batch_idx * intermedia_size + idx] = silu<T>(x) * y;
  }
}

template<>
__global__ void silu_and_mul_kernel<half>(
  half* out,               // [bs, intermedia size]
  const half* input,       // [bs, 2, intermedia size]
  const int intermedia_size) {
  const int batch_idx = blockIdx.x;
  // 获取fp16的向量大小
  int vec_size = Vec<half>::size;
  // 获取fp16的向量类型half2
  using Vec_t = typename Vec<half>::Type;
  for (int idx = threadIdx.x * vec_size; idx < intermedia_size; idx += blockDim.x) {
    // 与fp32实现的不同在于
    // 1.向量化读取
    // 2.使用hmul2向量化计算
    // 3.向量化写入
    const Vec_t x = *reinterpret_cast<Vec_t*>(const_cast<half*>(&input[batch_idx * 2 * intermedia_size + idx]));
    const Vec_t y = *reinterpret_cast<Vec_t*>(const_cast<half*>(&input[batch_idx * 2 * intermedia_size + intermedia_size + idx]));
    *reinterpret_cast<Vec_t*>(&out[batch_idx * intermedia_size + idx]) = __hmul2(silu<Vec_t>(x), y);
  }
}

template<typename T>
void launchAct(TensorWrapper<T>* input, TensorWrapper<T>* out) {
    int batch_size = input->shape[0];
    // 预防性检查，主要是防止shape的定义写错，导致不是我们expect的，那就比较难debug了
    LLM_CHECK(input->shape[1] == 2);
    int intermedia_size = input->shape[2];
    dim3 grid(batch_size);
    dim3 block(256);
    silu_and_mul_kernel<T><<<grid, block>>>(out->data, input->data, intermedia_size);
    // for debug，打印swiglu这个kernel的输出结果
#ifdef PRINT_DATA
    print_data<<<1, 1>>>(out->data);
#else
#endif
}
// We must instancite the template, if not, will report linking issue
template void launchAct(TensorWrapper<float>* input, TensorWrapper<float>* output);
template void launchAct(TensorWrapper<half>* input, TensorWrapper<half>* output);
