#include "hip/hip_runtime.h"
#include "src/kernels/attn_softmax_kernel.h"
#include "src/utils/cuda_debug_utils.cuh"
#include <float.h>
#include <math.h>
#include <assert.h>
#include <stdio.h>
// attention_score,    (batch_size, head_num, q_length, k_length), softmax output.
// qk,                 (batch_size, head_num, q_length, k_length), QK^T.
// attention_mask,     (batch_size, q_length, k_length), attention mask.
template <typename T>
struct SumOp
{
    __device__ __forceinline__ T operator()(const T &a, const T &b) const { return a + b; }
};

template <typename T>
struct MaxOp
{
    __device__ __forceinline__ T operator()(const T &a, const T &b) const { return max(a, b); }
};

template <template <typename> class ReductionOp, typename T>
__inline__ __device__ T warpReduce(T val)
{
    for (int mask = 32 / 2; mask > 0; mask /= 2)
    {
        val = ReductionOp<T>()(val, __shfl_xor_sync(0xffffffff, val, mask));
    }
    return val;
}

template <template <typename> class ReductionOp, typename T>
__inline__ __device__ T blockReduce(T val)
{
    int tid = threadIdx.x;
    int warp_id = tid / 32;
    int lane_id = tid % 32;
    int warp_nums = (blockDim.x + 31) / 32;
    static __shared__ T warp[64];
    val = warpReduce<ReductionOp, T>(val);
    if (lane_id == 0)
    {
        warp[warp_id] = val;
    }
    __syncthreads();
    float warp_val = tid < warp_nums ? warp[tid] : 0;
    return warpReduce<ReductionOp, T>(warp_val);
}

template <typename T, int NUMS_PER_THREAD_PER_ROW>
__global__ void ScaleMaskAndSoftmax_float(T *attn_score,
                                          T *qk,
                                          T *mask,
                                          int batch_size,
                                          int head_nums,
                                          int q_len,
                                          int k_len,
                                          float scale)
{
    int batch_id = blockIdx.y;
    int head_id = blockIdx.z;
    // note: NUMS_PER_THREAD_PER_ROW must be a constant value that known at compile time, following expr is invalid
    // const int NUMS_PER_THREAD_PER_ROW = ceil(k_len / blockDim.x);
    // warning: remember 1st priority thing is filtering the out-of-boundary threads
    if (threadIdx.x >= k_len)
    {
        return;
    }
    
    __shared__ float inv_sum, s_max;
    for (int row_start = blockIdx.x; row_start < q_len; row_start += gridDim.x)
    {
        int qk_offset = 0;
        int mask_offset = 0;
        T qk_data = static_cast<T>(0);
        T mask_data = static_cast<T>(0);
        T thread_max = FLT_MIN;
        // (Vincent)note: 面对这种一个block一个thread需要处理多行多列的时候，数据尽量用数组存储，计算出每个block和thread要处理几行几列
        T data[NUMS_PER_THREAD_PER_ROW]; 
        for (int col_start = 0; col_start < NUMS_PER_THREAD_PER_ROW; col_start++)
        {
            qk_offset = batch_id * head_nums * q_len * k_len + head_id * q_len * k_len + row_start * k_len + col_start * blockDim.x + threadIdx.x;
            qk_data = qk[qk_offset];

            mask_offset = batch_id * q_len * k_len + row_start * k_len + col_start * blockDim.x + threadIdx.x;
            mask_data = mask[mask_offset];
            // https://www.zhihu.com/question/472323371/answer/2001223766这
            data[col_start] = scale * qk_data + (1 - mask_data) * (-10000.0f);
            thread_max = fmax(data[col_start], thread_max);
        }
        // warp/block reduce
        T max_val = blockReduce<MaxOp, T>(thread_max);
        if (threadIdx.x == 0)
        {
            s_max = max_val;
            // debug info,printf("row max = %f\n", s_max);
        }
        __syncthreads();
        // thread local fenzi/fenmu
        T thread_sum = 0.0f;
        // for(int col_start = threadIdx.x; col_start < k_len; col_start += blockDim.x){
        for (int col_start = 0; col_start < NUMS_PER_THREAD_PER_ROW; col_start++)
        {
            qk_offset = batch_id * head_nums * q_len * k_len + head_id * q_len * k_len + row_start * k_len + col_start * blockDim.x + threadIdx.x;
            mask_offset = batch_id * q_len * k_len + row_start * k_len + col_start * blockDim.x + threadIdx.x;
            data[col_start] = expf(data[col_start] - s_max);
            thread_sum += data[col_start];
            // debug info,printf("after, data[%d]=%f, thread_sum = %f\n",col_start, data[col_start], thread_sum);
        }
        // row sum
        T sum = blockReduce<SumOp, T>(thread_sum);
        if (threadIdx.x == 0)
        {
            inv_sum = 1 / (sum + 1e-6f); // sum(fenmu) need to add a small value to avoid NAN
            // debug info, printf("row sum = %f\n", sum);
        }
        __syncthreads();
        // write back into gmem
        for (int col_start = 0; col_start < NUMS_PER_THREAD_PER_ROW; col_start++)
        {
            qk_offset = batch_id * head_nums * q_len * k_len + head_id * q_len * k_len + row_start * k_len + col_start * blockDim.x + threadIdx.x;
            attn_score[qk_offset] = (data[col_start] * inv_sum);
        }
    }
}
// cant partial specialize in func
template <typename T_half, int NUMS_PER_THREAD_PER_ROW>
__global__ void ScaleMaskAndSoftmax_half(T_half *attn_score,
                                         T_half *qk,
                                         T_half *mask,
                                         int batch_size,
                                         int head_nums,
                                         int q_len,
                                         int k_len,
                                         float scale)
{
    int batch_id = blockIdx.y;
    int head_id = blockIdx.z;
    // note: NUMS_PER_THREAD_PER_ROW must be a constant value that known at compile time, following expr is invalid
    // const int NUMS_PER_THREAD_PER_ROW = ceil(k_len / blockDim.x);
    int vec_size = Vec<T_half>::size;
    using Vec_t = typename Vec<T_half>::Type;

    Vec_t* attn_score_vec = reinterpret_cast<Vec_t*>(attn_score);
    Vec_t* qk_buf_vec = reinterpret_cast<Vec_t*>(qk);
    Vec_t* attn_mask_vec  = reinterpret_cast<Vec_t*>(mask);
    Vec_t ONE = scalar_cast_vec<Vec_t>(__float2half(1.0f));
    Vec_t NEG_INF = scalar_cast_vec<Vec_t>(__float2half(-10000.0f));
    Vec_t scale_vec = scalar_cast_vec<Vec_t>(__float2half(scale));

    __shared__ float inv_sum, s_max;
    // warning: remember 1st priority thing is filtering the out-of-boundary threads
    if (threadIdx.x * vec_size >= k_len)
    {
        return;
    }
    for (int row_start = blockIdx.x; row_start < q_len; row_start += gridDim.x)
    {
        int qk_offset = 0;
        int mask_offset = 0;
        Vec_t qk_data;
        Vec_t mask_data;
        float thread_max = FLT_MIN;
        Vec_t data[NUMS_PER_THREAD_PER_ROW]; // 面对这种一个block一个thread需要处理多行多列的时候，数据尽量用数组存储，计算出每个block和thread要处理几行几列
        // for(int col_start = threadIdx.x; col_start < k_len; col_start += blockDim.x){
        for (int col_start = 0; col_start < NUMS_PER_THREAD_PER_ROW; col_start++)
        {
            qk_offset = batch_id * head_nums * q_len * k_len / 2 + head_id * q_len * k_len / 2  + row_start * k_len / 2 + col_start * blockDim.x + threadIdx.x;
            qk_data = qk_buf_vec[qk_offset];

            mask_offset = batch_id * q_len * k_len / 2 + row_start * k_len / 2 + col_start * blockDim.x + threadIdx.x;
            mask_data = attn_mask_vec[mask_offset];
            Vec_t mask_vec_reg= __hmul2(__hsub2(ONE, mask_data), NEG_INF);

            data[col_start] = __hadd2(__hmul2(scale_vec, qk_data), mask_vec_reg);
            //printf("after,scale=%f, qk_data=%f, qk_offset=%d, tid=%d,  mask_data=%f,data[%d]=%f\n",(float)scale_vec.x, (float)qk_data.x, qk_offset, threadIdx.x, (float)mask_data.x, col_start, (float)data[col_start].x);
            thread_max = fmax(fmax((float)data[col_start].x, (float)data[col_start].y), thread_max);
        }
        // warp/block reduce
        float max_val = blockReduce<MaxOp, float>(thread_max);
        if (threadIdx.x == 0)
        {
            s_max = max_val;
            //printf("row max = %f\n", s_max);
        }
        __syncthreads();
        // thread local fenzi/fenmu
        float thread_sum = 0.0f;
        // for(int col_start = threadIdx.x; col_start < k_len; col_start += blockDim.x){
        for (int col_start = 0; col_start < NUMS_PER_THREAD_PER_ROW; col_start++)
        {
            // debug info to see useless threads if its available,printf("blockIdx.x=%d, threadIdx.x=%d\n",blockIdx.x, threadIdx.x);
            data[col_start] = h2exp(__hsub2(data[col_start], scalar_cast_vec<Vec_t>(s_max)));
            thread_sum += (float)(__hadd(data[col_start].x, data[col_start].y));
            // debug info,printf("after, data[%d]=%f, thread_sum = %f\n",col_start, data[col_start], thread_sum);
        }
        // row sum
        float sum = blockReduce<SumOp, float>(thread_sum);
        if (threadIdx.x == 0)
        {
            inv_sum = 1 / (sum + 1e-6f); // sum(fenmu) need to add a small value to avoid NAN
            //printf("row sum = %f\n", sum);
        }
        __syncthreads();
        // write back into gmem
        for (int col_start = 0; col_start < NUMS_PER_THREAD_PER_ROW; col_start++)
        {
            qk_offset = batch_id * head_nums * q_len * k_len / 2 + head_id * q_len * k_len / 2 + row_start * k_len / 2 + col_start * blockDim.x + threadIdx.x;
            attn_score_vec[qk_offset] = __hmul2(data[col_start], scalar_cast_vec<Vec_t>(inv_sum));
        }
    }
}

#define LAUNCH_SOFTMAX(dtype, vec_size)                                                                         \
    if (block.x > 2048 && block.x <= 4096)                                                                      \
    {                                                                                                           \
        constexpr int NUMS_PER_THREAD_PER_ROW = 4;                                                              \
        block.x /= 4 * vec_size;                                                                                \
        block.x = (block.x + 32 - 1) / 32 * 32;                                                                 \
        assert(block.x < 1024);                                                                                 \
        ScaleMaskAndSoftmax_##dtype<dtype, NUMS_PER_THREAD_PER_ROW><<<grid, block>>>((dtype *)attn_score->data, \
                                                                                     (dtype *)qk->data,         \
                                                                                     (dtype *)mask->data,       \
                                                                                     batch_size,                \
                                                                                     head_nums,                 \
                                                                                     q_length,                  \
                                                                                     k_length,                  \
                                                                                     scale);                    \
    }                                                                                                           \
    else if (block.x > 1024)                                                                                    \
    {                                                                                                           \
        constexpr int NUMS_PER_THREAD_PER_ROW = 2;                                                              \
        block.x /= 2 * vec_size;                                                                                \
        \                                    
        block.x = (block.x + 32 - 1) / 32 * 32;                                                                 \
        \   
        assert(block.x < 1024);                                                                                 \
        ScaleMaskAndSoftmax_##dtype<dtype, NUMS_PER_THREAD_PER_ROW><<<grid, block>>>((dtype *)attn_score->data, \
                                                                                     (dtype *)qk->data,         \
                                                                                     (dtype *)mask->data,       \
                                                                                     batch_size,                \
                                                                                     head_nums,                 \
                                                                                     q_length,                  \
                                                                                     k_length,                  \
                                                                                     scale);                    \
    }                                                                                                           \
    else                                                                                                        \
    {                                                                                                           \
        \ 
        constexpr int NUMS_PER_THREAD_PER_ROW = 1;                                                              \
        block.x /= vec_size;                                                                                    \
        assert(block.x < 1024);                                                                                 \
        ScaleMaskAndSoftmax_##dtype<dtype, NUMS_PER_THREAD_PER_ROW><<<grid, block>>>((dtype *)attn_score->data, \
                                                                                     (dtype *)qk->data,         \
                                                                                     (dtype *)mask->data,       \
                                                                                     batch_size,                \
                                                                                     head_nums,                 \
                                                                                     q_length,                  \
                                                                                     k_length,                  \
                                                                                     scale);                    \
        \      
                                                                                                     \
    }

template <typename T>
void launchScaleMaskAndSoftmax(TensorWrapper<T> *qk,
                               TensorWrapper<T> *mask,
                               TensorWrapper<T> *attn_score,
                               float scale)
{
    // attention_score,    (batch_size, head_num, q_length, k_length), softmax output.
    // qk,                 (batch_size, head_num, q_length, k_length), QK^T.
    // attention_mask,     (batch_size, q_length, k_length), attention mask.
    int q_length = qk->shape[2];
    int batch_size = qk->shape[0];
    int head_nums = qk->shape[1];
    int k_length = qk->shape[3];
    bool is_half = sizeof(T) == 2;
    // TODO: should enhance it by padding to support odd ones
    if (is_half) {
    	LLM_CHECK_WITH_INFO(k_length % 2 == 0, "Currently, K_len should be divided by 2 under half type!");
    }
    dim3 grid(q_length, batch_size, head_nums);
    dim3 block((k_length + 32 - 1) / 32 * 32); // align with 32x threads
    if (is_half)
    {
        LAUNCH_SOFTMAX(half, 2);
    }
    else
    {
        LAUNCH_SOFTMAX(float, 1);
    }
#ifdef PRINT_DATA
    print_data<<<1, 1>>>(attn_score->data);
#else
#endif
}

template void launchScaleMaskAndSoftmax(TensorWrapper<float> *qk,
                                        TensorWrapper<float> *mask,
                                        TensorWrapper<float> *attn_score,
                                        float scale);

template void launchScaleMaskAndSoftmax(TensorWrapper<half> *qk,
                                        TensorWrapper<half> *mask,
                                        TensorWrapper<half> *attn_score,
                                        float scale);
