#include "hip/hip_runtime.h"
#include "src/kernels/build_casual_mask.h"
// 此算子仅使用在context decoder阶段，用于遮盖掉seq当前位置之后的信息，防止模型使用未来的信息
// 而self decoder是一个自回归模型，本就没有未来的信息
// mask shape =  [bs, max_q_len, max_k_len]
template<typename T>
__global__ void BuildCausalMasksConsideringContextPastKV(T* mask,
                                                const int* q_lens,  //input lens, shape=[batch size]
                                                const int* k_lens,  //context lens, shape=[batch size]
                                                int max_q_len,  // max(q_lens)
                                                int max_k_len){ // max(k_lens)
    int tid = threadIdx.x;
    // 核函数共分配了bs个block，可以方便得通过block id来访问q_lens和k_lens数组中的值
    // 一个block负责处理一个bs大小中的数
    int qlen = q_lens[blockIdx.x];
    int klen = k_lens[blockIdx.x];
    // 偏移一个bs大小的空间
    // 即blockIdx.x==0时，指向mask数组的开头；blockIdx.x==1时，指向mask数组偏移了max_q_len * max_k_len大小后的位置
    mask += blockIdx.x * max_q_len * max_k_len;
    // offset用于表示每个bs内部的偏移
    int offset = threadIdx.x;
    // note: this judgement confirms we dont exceed data boundry
    while (offset < max_q_len * max_k_len){
        // 分别求出行号q和列号k
        int q = offset / max_k_len;
        int k = offset % max_k_len;
        // 此处与视频中的代码不同，k考虑了多轮对话的上下文序列，但设置mask时 k >= klen - qlen 将旧序列一并遮去了
        // 下图为支持多轮对话的mask
        // 1 1 1 | 1 -inf -inf
        // 1 1 1 | 1   1  -inf
        // 1 1 1 | 1   1    1
        // 下图为不支持多轮对话的mask
        // -inf -inf -inf | 1 -inf -inf
        // -inf -inf -inf | 1   1  -inf
        // -inf -inf -inf | 1   1    1
        // "|"符号前表示旧的对话序列，符号后表示当前轮的对话序列
        bool is_one = q < qlen && k < klen && k <= q + (klen - qlen) && k >= klen - qlen;
        mask[offset] = static_cast<T>(is_one);

        // 保证遍历完一个bs中所有的空间
        offset += blockDim.x;
    }
}

template<typename T>
void launchBuildCausalMasks(TensorWrapper<T>* mask, 
                            TensorWrapper<int>* q_lens, 
                            TensorWrapper<int>* k_lens)
{
    int batch_size = mask->shape[0];
    int max_q_len = mask->shape[1];
    int max_k_len = mask->shape[2];
    // 此处的max_q_len和max_k_len是经过统计后得出的外部输入
    BuildCausalMasksConsideringContextPastKV<T><<<batch_size, 256>>>(mask->data, q_lens->data, k_lens->data, max_q_len, max_k_len);
}

template void launchBuildCausalMasks(TensorWrapper<float>* mask, 
                            TensorWrapper<int>* q_lens, 
                            TensorWrapper<int>* k_lens);

template void launchBuildCausalMasks(TensorWrapper<half>* mask, 
                            TensorWrapper<int>* q_lens, 
                            TensorWrapper<int>* k_lens);
