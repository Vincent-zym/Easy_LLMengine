#include "hip/hip_runtime.h"
#include "src/kernels/build_casual_mask.h"
// mask shape =  [bs, max_q_len, max_k_len]
template<typename T>
__global__ void BuildCausalMasksConsideringContextPastKV(T* mask,
                                                const int* q_lens,  //input lens, shape=[batch size]
                                                const int* k_lens,  //context lens, shape=[batch size]
                                                int max_q_len,  // max(q_lens)
                                                int max_k_len){ // max(k_lens)
    int tid = threadIdx.x;
    int qlen = q_lens[blockIdx.x];
    int klen = k_lens[blockIdx.x];
    mask += blockIdx.x * max_q_len * max_k_len;
    int offset = threadIdx.x;
    // note: this judgement confirms we dont exceed data boundry
    while (offset < max_q_len * max_k_len){
        // 分别求出行号q和列号k
        int q = offset / max_k_len;
        int k = offset % max_k_len;
        // 此处与视频中的代码不同，k考虑了多轮对话的上下文序列，但设置mask时 k >= klen - qlen 将旧序列一并遮去了
        // 下图为支持多轮对话的mask
        // 1 1 1 | 1 -inf -inf
        // 1 1 1 | 1   1  -inf
        // 1 1 1 | 1   1    1
        // 下图为不支持多轮对话的mask
        // -inf -inf -inf | 1 -inf -inf
        // -inf -inf -inf | 1   1  -inf
        // -inf -inf -inf | 1   1    1
        // "|"符号前表示旧的对话序列，符号后表示当前轮的对话序列
        bool is_one = q < qlen && k < klen && k <= q + (klen - qlen) && k >= klen - qlen;
        mask[offset] = static_cast<T>(is_one);

        offset += blockDim.x;
    }
}

template<typename T>
void launchBuildCausalMasks(TensorWrapper<T>* mask, 
                            TensorWrapper<int>* q_lens, 
                            TensorWrapper<int>* k_lens)
{
    int batch_size = mask->shape[0];
    int max_q_len = mask->shape[1];
    int max_k_len = mask->shape[2];
    BuildCausalMasksConsideringContextPastKV<T><<<batch_size, 256>>>(mask->data, q_lens->data, k_lens->data, max_q_len, max_k_len);
}

template void launchBuildCausalMasks(TensorWrapper<float>* mask, 
                            TensorWrapper<int>* q_lens, 
                            TensorWrapper<int>* k_lens);

template void launchBuildCausalMasks(TensorWrapper<half>* mask, 
                            TensorWrapper<int>* q_lens, 
                            TensorWrapper<int>* k_lens);
