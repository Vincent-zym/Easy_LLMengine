#include "hip/hip_runtime.h"
#include "src/kernels/cal_paddingoffset.h"
// shape:
    //seq_lengths:[batch size]
    //cum_seqlens:[batch size + 1],first ele is 0
    //padding_offset:[batch size * max q len]
// note: the point is to calc padding offset and cum offset

// 例子 bs = 3, seqlen = [3,2,5], max_seq_len = 5
// 1 1 1 0 0
// 1 1 0 0 0
// 1 1 1 1 1
// paddingoffset 为
// 0 0 0 0 0
// 2 2 2 2 2
// 5 5 5 5 5
__global__ void CalPaddingoffset(int*         padding_offset, 
                                int*         cum_seqlens,
                                const int*   input_lengths, //actual input lens
                                const int    batch_size,
                                const int    max_q_len) {
    int ind = 0;
    int cum_offset = 0;
    int total_seqlen = 0;
    // 遍历每一个批次
    for(int b = 0; b < batch_size; b++) {
        // 获取到每个句子的长度
        int seqlen = input_lengths[b];
        // 累计的句子长度
        cum_seqlens[b] = total_seqlen;
        // 遍历一个句子里的所有token位置
        // each token in one seq has same cum offset
        for (int i = 0; i < seqlen; i++) {
            // index是对于每个token的索引，每个token都有一个paddingoffset
            padding_offset[ind] = cum_offset;
            ind++;
        }
        // 获取累计的 padding offset 和 总共的句子长度
        cum_offset += max_q_len - seqlen;
        total_seqlen += seqlen;
    }
    // 注意 cum_seqlens 的形状，添加最后一个累计句子长度（总长度）
    cum_seqlens[batch_size] = total_seqlen;
}

// 这个函数的目的是为了在attention之后，可以方便的移除padding。
// padding操作和 seq len 维度相关，因此相关操作需要在不涉及这一维度的计算后添加。  
void launchCalPaddingoffset(TensorWrapper<int>* padding_offset, 
                            TensorWrapper<int>* cum_seqlens,
                            TensorWrapper<int>* input_lengths)//actual input lens
{
    const int batch_size = padding_offset->shape[0];                            
    const int max_q_len = padding_offset->shape[1]; 
    LLM_CHECK_WITH_INFO(batch_size == input_lengths->shape[0], "input lenghts numbers should equal to padding offset bs dim!") ;                        
    LLM_CHECK_WITH_INFO(batch_size == cum_seqlens->shape[0] - 1, "cum seqlen numbers should equal to padding offset bs dim + 1!") ;                        
    CalPaddingoffset<<<1, 1>>>( 
        padding_offset->data, cum_seqlens->data, input_lengths->data, batch_size, max_q_len
    );
}