#include "hip/hip_runtime.h"
#include <iostream>
#include "src/utils/cuda_debug_utils.cuh"
#include "src/kernels/fused_transpose_and_remv_pad.h"
// [bs,head nums,seqlen,head size]=>[bs,seqlen,head nums,head size]=>[num tokens,head nums,head size]
// padding_offset.shape = [num_tokens]
template <typename T>
__global__ void fused_transpose_reshape_remv_pad(T *src,
                                                 T *dst,
                                                 const int num_tokens,
                                                 const int batch_size,
                                                 const int seq_len,
                                                 const int head_num,
                                                 const int head_size,
                                                 const int *padding_offset /*for remove padding*/)
{
    int token_id = blockIdx.x; 
    // map to input id
    int batch_id = (blockIdx.x + padding_offset[token_id]) / seq_len;
    int seq_id = (blockIdx.x + padding_offset[token_id]) % seq_len;
    // compute the offset of transpose and remove padding before or after
    int src_offset = batch_id * head_num * seq_len * head_size + seq_id * head_size;
    int dst_offset = token_id * head_num * head_size;

    for (int i = threadIdx.x; i < head_num * head_size; i += blockDim.x)
    {
        int head_id = i / head_size;
        int head_size_id = i % head_size;
        dst[dst_offset + i] = src[src_offset + head_id * seq_len * head_size + head_size_id];
    }
}
template <typename T>
void launchTransposeOutRemovePadding(TensorWrapper<T> *qkv_buf_w_pad,
                                     TensorWrapper<int> *padding_offset,
                                     TensorWrapper<T> *qkv_buf_wo_pad_1)
{
    int batch_size = qkv_buf_w_pad->shape[0];
    int head_num = qkv_buf_w_pad->shape[1];
    int seq_len = qkv_buf_w_pad->shape[2];
    int head_size = qkv_buf_w_pad->shape[3];
    int num_tokens = qkv_buf_wo_pad_1->shape[0];
    dim3 grid(num_tokens);
    dim3 block(std::min(head_num * head_size, 1024));
    fused_transpose_reshape_remv_pad<T><<<grid, block>>>(qkv_buf_w_pad->data,
                                                         qkv_buf_wo_pad_1->data,
                                                         num_tokens,
                                                         batch_size,
                                                         seq_len,
                                                         head_num,
                                                         head_size,
                                                         padding_offset->data);
#ifdef PRINT_DATA
    print_data<<<1, 1>>>(qkv_buf_wo_pad_1->data);
#else
#endif
}

template void launchTransposeOutRemovePadding(TensorWrapper<float> *qkv_buf_w_pad,
                                              TensorWrapper<int> *padding_offset,
                                              TensorWrapper<float> *qkv_buf_wo_pad_1);
template void launchTransposeOutRemovePadding(TensorWrapper<half> *qkv_buf_w_pad,
                                              TensorWrapper<int> *padding_offset,
                                              TensorWrapper<half> *qkv_buf_wo_pad_1);
