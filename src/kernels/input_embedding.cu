#include "hip/hip_runtime.h"
#include <stdio.h>
#include "src/kernels/input_embedding.h"
//#include "src/utils/cuda_debug_utils.cuh"
template<typename T>
__global__ void embeddingFunctor(const int* input_ids,
               T* output, 
               const T* embed_table,
               const int max_context_token_num,
               const int hidden_size)
{
    //得到全局线程id
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    // 如果分配的总线程数不够，那么在输出范围内的线程id再来多轮
    while (index < max_context_token_num * hidden_size) {
        // 拿到token_num这一个维度的序号，输出的行号
        int id = input_ids[index / hidden_size];
        // 通过token id索引到对应embedding table的行号，此行号乘上embedding的列数，得到该token id的hidden units
        // 每个线程并行地把hidden size个值读取并写到对应线程id位置的output
        output[index] = embed_table[id * hidden_size + index % hidden_size];
        // 当前线程处理完一轮，累加index到下一轮，防止总线程数不够。
        index += blockDim.x * gridDim.x;
    }
}

template<typename T>
void launchInputEmbedding(TensorWrapper<int>* input_ids,    // INT [token num]
                          TensorWrapper<T>* output,       // FP32 [token num, hidden_size] = [token num, 4096]
                          EmbeddingWeight<T>* embed_table// FP32 [vocal_size, hidden_size]
                          ) {
    //分配线程块，核函数需要的维度信息 
    const int blockSize = 256;
    const int max_context_token_num = output->shape[0]; // token num
    const int hidden_size = output->shape[1];
    const int gridSize = 2048;
    LLM_CHECK_WITH_INFO(max_context_token_num == input_ids->shape[0], "input ids 1st shape should equal to 1st shape of output");
    embeddingFunctor<T><<<gridSize, blockSize>>>(input_ids->data,
                                                 output->data,
                                                 embed_table->data,
                                                 max_context_token_num,
                                                 hidden_size);
#ifdef PRINT_DATA
    print_data<<<1, 1>>>(output->data);
#else
#endif
}

// 显式实例化模版函数，由于cuda的语法规则，不能存在.cpp文件里，因此只能在此实例化
template void launchInputEmbedding(TensorWrapper<int>* input_ids,    
                                   TensorWrapper<float>* output,       
                                   EmbeddingWeight<float>* embed_table);
template void launchInputEmbedding(TensorWrapper<int>* input_ids,    
                                   TensorWrapper<half>* output,       
                                   EmbeddingWeight<half>* embed_table);
