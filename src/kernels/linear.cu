#include "hip/hip_runtime.h"
#include <iostream>
#include <fstream>
#include "src/utils/cuda_debug_utils.cuh"
#include "src/kernels/linear.h"
// TODO: when abstracted weight class, replace T with class
// all matmul cases:
// ctx qkv lienar: [num_tokens, qhiddenunits] * [qhiddenunits, hiddenunits] = {num_tokens, qkv_head_num,  head_size}
// ctx attn output linear: {num_tokens, head_num, head_size} * {q hidden units, q hidden units} = {num_tokens, q hidden units}
// self qkv linear: [bs, q hidden units] * [qhiddenunits, hiddenunits] = {bs, qkv_head_num,  head_size}}
// self attn output linear: {batch_size, q hidden_units} * [qhiddenunits, qhiddenunits] = [bs, q hiddenunits]
// lmhead linear: [bs, q hidden units] * [vocab size, q hiden units], need transpose B
// gate:[bs/token nums, q hidden units] * [q hidden units, inter size] = [bs/token nums, inter size]
// up:[bs/token nums, q hidden units] * [q hidden units, inter size] = [bs/token nums, inter size]
// fusedGateUpGemm: [bs/token nums, q hidden units] * [q hidden units, 2 * inter size] = [bs/token nums, 2 * inter size]
// down:[bs/token nums, inter size] * [q hidden units, inter size] = [bs/token nums, q hidden units]
template <typename T>
void launchLinearGemm(TensorWrapper<T> *input,
                      BaseWeight<T> &weight,
                      TensorWrapper<T> *output,
                      cublasWrapper *cublas_wrapper,
                      bool trans_a,
                      bool trans_b)
{
    int Am = weight.shape[1];
    int Ak = weight.shape[0];
    int Bk = input->shape[1];
    int Bn = input->shape[0];
    int Cm = output->shape[1];
    int Cn = output->shape[0];
    // for ctx attn and self attn qkv linear, assume [bs/token nums, qkv h ead num, head size]
    // for gate & up linear, assume weight.shape=[hidden,2*intersize], output.shape=[bs, 2, inter size]
    Cm = output->shape.size() == 3 ? output->shape[1] * output->shape[2] : output->shape[1];
    // for ctx attn output linear
    Bk = input->shape.size() == 3 ? input->shape[1] * input->shape[2] : input->shape[1];
    int lda = Am;
    int ldb = Bk;
    int ldc = Cm;

    // for lmhead linear and ffn all lieanrs
    hipblasOperation_t transA = trans_b ? HIPBLAS_OP_T : HIPBLAS_OP_N;
    hipblasOperation_t transB = trans_a ? HIPBLAS_OP_T : HIPBLAS_OP_N;
    if (!trans_a && !trans_b)
    {
        LLM_CHECK_WITH_INFO(Ak == Bk, "2nd dim of input MUST = 1st dim of weight");
    }
    cublas_wrapper->Gemm(transA,
                         transB,
                         trans_b ? Ak : Am, // m
                         Cn,                // n, when load real weight, lmhead weight is same as pre embedding, which shape = [vocab, hidden], so here should transpose b
                         Bk,
                         weight.data,  // A, cur_input_len is for context decoder lmhead
                         lda,          // lda
                         input->data,  // B
                         ldb,          // ldb
                         output->data, // C
                         ldc,          // ldc
                         1.0f,
                         0.0f);
#ifdef PRINT_DATA
    print_data<<<1, 1>>>(output->data);
#else
#endif
}

template <typename T>
void launchLinearStridedBatchGemm(TensorWrapper<T> *input1,
                                  TensorWrapper<T> *input2,
                                  TensorWrapper<T> *output,
                                  cublasWrapper *cublas_wrapper,
                                  bool trans_a,
                                  bool trans_b)
{
    // B.T A.T = C.T
    // TODO:currently only consider trans_b
    int Bm = input1->shape[2]; // len q       // len q
    int Bk = input1->shape[3]; // head size   // len k
    int Ak = input2->shape[2]; // len k       // len k
    int An = input2->shape[3]; // head size   // head size
    int Cm = output->shape[2]; // len q       // len q
    int Cn = output->shape[3]; // len k       // head size
    int lda = An;
    int ldb = Bk; // ld should be val before transpose
    int ldc = Cn;
    int64_t strideA = Ak * An; // stride should be val after transpose
    int64_t strideB = Bm * Bk;
    int64_t strideC = Cm * Cn;
    // TODO:check 4nd dim of input = 3rd dim of weight
    // TODO:check batchCount of two matrix is equal
    int batchCount = input1->shape[0] * input1->shape[1];

    hipblasOperation_t transA = trans_b ? HIPBLAS_OP_T : HIPBLAS_OP_N;
    hipblasOperation_t transB = trans_a ? HIPBLAS_OP_T : HIPBLAS_OP_N;

    cublas_wrapper->stridedBatchedGemm(transA,
                                       transB,
                                       Cn,           // m
                                       Cm,           // n
                                       Bk,           // k
                                       input2->data, // A,[Bk, Bn]=[bs, head num,  head size,max k len]
                                       lda,
                                       strideA,
                                       input1->data, // B [Ak, An]=[bs, head num,  head size,max q len]
                                       ldb,
                                       strideB,
                                       output->data, // C [[bs, head num,  max k len, max q len]
                                       ldc,
                                       strideC,
                                       batchCount,
                                       1.0f,
                                       0.0f);
#ifdef PRINT_DATA
    print_data<<<1, 1>>>(output->data);
#else
#endif
}

template void launchLinearGemm(TensorWrapper<float> *input,
                               BaseWeight<float> &weight,
                               TensorWrapper<float> *output,
                               cublasWrapper *cublas_wrapper,
                               bool trans_a,
                               bool trans_b);

template void launchLinearGemm(TensorWrapper<half> *input,
                               BaseWeight<half> &weight,
                               TensorWrapper<half> *output,
                               cublasWrapper *cublas_wrapper,
                               bool trans_a,
                               bool trans_b);

template void launchLinearStridedBatchGemm(TensorWrapper<float> *input1,
                                           TensorWrapper<float> *input2,
                                           TensorWrapper<float> *output,
                                           cublasWrapper *cublas_wrapper,
                                           bool trans_a,
                                           bool trans_b);

template void launchLinearStridedBatchGemm(TensorWrapper<half> *input1,
                                           TensorWrapper<half> *input2,
                                           TensorWrapper<half> *output,
                                           cublasWrapper *cublas_wrapper,
                                           bool trans_a,
                                           bool trans_b);
