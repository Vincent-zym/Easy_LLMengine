#include "hip/hip_runtime.h"
#include <algorithm>   // std::fill_n
#include <iostream>    // snprintf
#include <math.h>      // expf, log
#include <stdlib.h>    // rand
#include <string>      // std::string
#include <vector>      // std::vector
#include "src/kernels/act_kernel.h"
// (Vincent)note: not sure CPU implementation is absolutely right and the GPU kernel is right compared with HF.
// when you are implementing LLMs inference on CPU, you can reuse the CPU kernel and test its correctness
// (Vincent)note:
// `./test_act 1` to test half GPU kernel
// `./test_act` to test fp32 GPU kernel
template<typename T>
void CPUSwiGLU(T* input, T* output, int batch_size, int intermedia_size){
    float silu_out = 0.0f;
    for(int batch_id = 0; batch_id < batch_size; batch_id++){
        for(int i = 0; i < intermedia_size; i++) {
            int offset1 = batch_id * 2 * intermedia_size + i;
            int offset2 = batch_id * 2 * intermedia_size + i + intermedia_size;
            int out_offset = batch_id * intermedia_size + i;
            silu_out = (float)input[offset1] / (1.0f + expf(-1 * (float)input[offset1]));
            output[out_offset] = static_cast<T>(silu_out * (float)input[offset2]);
        }
    }
}
template<typename T>
bool CheckResult(T* CPUoutput, T* GPUoutput, int output_size) {
    for(int i = 0; i < output_size; i++) {
        if(fabs((float)CPUoutput[i] - (float)GPUoutput[i]) > 1e-6){
	    printf("the %dth res is wrong, CPUoutput = %f, GPUoutput = %f\n", i, (float)CPUoutput[i], (float)GPUoutput[i]);
        }
    }
    return true;
}

template<typename T>
void test_act(int batch_size, int intermedia_size, int input_size , int output_size) {
    T* h_input;
    T* d_input;
    h_input = (T*)malloc(sizeof(T) * input_size);
    hipMalloc((void**)&d_input, sizeof(T) * input_size);
    T* h_output;
    T* d_output;
    h_output = (T*)malloc(sizeof(T) * output_size);
    hipMalloc((void**)&d_output, sizeof(T) * output_size);
    for(int i = 0; i < input_size; i++) { // initialize host data
        h_input[i] = (T)1;
    }
    hipMemcpy(d_input, h_input, sizeof(T) * input_size, hipMemcpyHostToDevice);
    DataType type = getTensorType<T>();
    TensorWrapper<T>* input_tensor = new TensorWrapper<T>(GPU, type, {batch_size, 2, intermedia_size}, d_input);
    TensorWrapper<T>* output_tensor = new TensorWrapper<T>(GPU, type, {batch_size, intermedia_size}, d_output);
    launchAct(input_tensor, output_tensor);
    hipMemcpy(h_output, d_output, sizeof(T) * output_size, hipMemcpyDeviceToHost);
    T* CPU_output = (T*)malloc(sizeof(T) * output_size);
    CPUSwiGLU(h_input, CPU_output, batch_size, intermedia_size);
    bool is_true = CheckResult(CPU_output, h_output, output_size);
    if(is_true){
        printf("test passed");
    } else {
        printf("test failed");
    }

    free(h_input);
    free(h_output);
    free(CPU_output);
    hipFree(d_input);
    hipFree(d_output);    
}

int main(int argc, char** argv) {
    constexpr int batch_size = 16;
    constexpr int intermedia_size = 11008;
    constexpr int input_size = batch_size * intermedia_size * 2;
    constexpr int output_size = batch_size * intermedia_size;
    if (argv[1]){
        test_act<half>(batch_size, intermedia_size, input_size, output_size);
    } else {
        test_act<float>(batch_size, intermedia_size, input_size, output_size);
    }
    
}
