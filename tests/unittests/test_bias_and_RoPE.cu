#include "hip/hip_runtime.h"
#include <algorithm>   // std::fill_n
#include <iostream>    // snprintf
#include <math.h>      // expf, log
#include <stdlib.h>    // rand
#include <string>      // std::string
#include <vector>      // std::vector

#include "src/kernels/qkv_bias_and_RoPE.h"
#include "src/weights/llama/attention_weights.h"
#include "src/utils/macro.h"
// (RussWong)note: not sure CPU implementation is absolutely right and the GPU kernel is right compared with HF.
// when you are implementing LLMs inference on CPU, you can reuse the CPU kernel and test its correctness
void CPUfunc(float* q,
                float* k,
                float* v,
                float* QKV,
                const float* qkv_bias,
                const int*   padding_offset,
                const int*   history_length,
                const int*   input_length,
                const int    batch_size,
                const int    seq_len,
                const int    token_num,
                const int    head_num,
                const int    kv_head_num,
                const int    head_size,
                const int    rotary_embedding_dim,
                float        rotary_embedding_base) {
    int qbatchstride = seq_len * head_num * head_size;
    int kvbatchstride = seq_len * kv_head_num * head_size;
    for (int b = 0; b < batch_size; b++) {
        for (int s = 0; s < seq_len; s++) {
            int timestep = history_length[b] + s;
            for (int head = 0; head < head_num; head++) {
                for (int d = 0; d < head_size; d++) {
                    //q bias
                    q[b * qbatchstride + s * head_num * head_size + head * head_size + d] = 
                            QKV[b * qbatchstride + s * head_num * head_size + head * head_size + d];
                }
                   //RoPE
                for (int d = 0; d < head_size / 2; d++) {
                    float x0 = q[b * qbatchstride + s * head_num * head_size + head * head_size + d];
                    float x1 = q[b * qbatchstride + s * head_num * head_size + head * head_size + d + 64];
                    // refer to https://zhuanlan.zhihu.com/p/647109286, d=0,2,4,dim-1
                    float inv_freq = timestep / powf(rotary_embedding_base, (d * 2) / (float)rotary_embedding_dim);
                    q[b * qbatchstride + s * head_num * head_size + head * head_size + d] = 
                                                                        x0 * cos(inv_freq) - x1 * sin(inv_freq);
                    
                    q[b * qbatchstride + s * head_num * head_size + head * head_size + d + 64] = 
                                                                        x1 * cos(inv_freq) + x0 * sin(inv_freq);

                } 
            }
            for (int head = 0; head < kv_head_num; head++) {
                for (int d = 0; d < head_size; d++) {
                    //k bias
                    k[b * kvbatchstride + s * kv_head_num * head_size + head * head_size + d] = 
                            QKV[b * kvbatchstride + s * (head_num + kv_head_num) * head_size + head * head_size + d];// + qkv_bias[(head_num + kv_head_num)  * head_size + d];
                    v[b * kvbatchstride + s * kv_head_num * head_size + head * head_size + d] = 
                            QKV[b * kvbatchstride + s * (head_num + kv_head_num * 2) * head_size + head * head_size + d];// + qkv_bias[(head_num + 2 * kv_head_num)  * head_size + d];
                }
                   //RoPE
                for (int d = 0; d < head_size / 2; d++) {
                    float x0 = k[b * kvbatchstride + s * kv_head_num * head_size + head * head_size + d];
                    float x1 = k[b * kvbatchstride + s * kv_head_num * head_size + head * head_size + d + 64];
                    float inv_freq = timestep / powf(rotary_embedding_base, (d * 2) / (float)rotary_embedding_dim);
                    k[b * kvbatchstride + s * kv_head_num * head_size + head * head_size + d] = 
                                                                        x0 * cos(inv_freq) - x1 * sin(inv_freq);
                    
                    k[b * kvbatchstride + s * kv_head_num * head_size + head * head_size + d + 64] = 
                                                                        x1 * cos(inv_freq) + x0 * sin(inv_freq);

                } 
            }            
        }
    }
}

bool CheckResult(float* q, float* k, float* hq, float* hk, 
                const int q_size, const int k_size) {
    for(int i = 0; i < q_size; i++) {
        if(fabs(q[i] - hq[i]) > 1e-6){
            printf("the %dth q is wrong, q = %f, hq = %f\n", i, q[i], hq[i]);
            return false;
        }
    }
    for(int i = 0; i < k_size; i++) {
        if(fabs(k[i] - hk[i]) > 1e-6){
            printf("the %dth k is wrong, k = %f, hk = %f\n", i, k[i], hk[i]);
            return false;
        }
    }
    return true;
}
// (RussWong)note:
// `./biasRope` to test fp32 GPU kernel
// half GPU kernel test is not implemented now
int main() {
    const int batch_size = 1;
    const int seq_len = 32;
    int* padding_offset = (int*)malloc(sizeof(int) * batch_size * seq_len);
    int* history_length = (int*)malloc(sizeof(int) * batch_size);
    int* input_length = (int*)malloc(sizeof(int) * batch_size);
    const int token_num = batch_size * seq_len;
    const int head_num = 32;
    const int kv_head_num = 32;
    const int head_size = 128;
    const int rotary_embedding_dim = 128;
    const int rotary_embedding_base = 10000;
    const int max_position_embeddings = 2048;
    
    float* q = (float*)malloc(sizeof(float) * batch_size * seq_len * head_num * head_size); //output
    float* k = (float*)malloc(sizeof(float) * batch_size * seq_len * kv_head_num * head_size); //output
    float* v = (float*)malloc(sizeof(float) * batch_size * seq_len * kv_head_num * head_size); //output
    float* QKV = (float*)malloc(sizeof(float) * token_num * (head_num + 2 * kv_head_num) * head_size);
    float* qkv_bias = (float*)malloc(sizeof(float) * (head_num + 2 * kv_head_num) * head_size);
    for(int i = 0; i < token_num * (head_num + 2 * kv_head_num) * head_size; i++){
        QKV[i] = 32.0f;
    }
    for(int i = 0; i < (head_num + 2 * kv_head_num) * head_size; i++){
        qkv_bias[i] = 2.0f;
    }
    for(int i = 0; i < batch_size; i++){
	input_length[i] = 7;
        history_length[i] = 0;
    }
    for(int i = 0; i < batch_size * seq_len; i++){
        padding_offset[i] = 0;
    }

    int* dpadding_offset;
    int* dhistory_length; 
    int* dinput_length;
    float* dq;
    float* dk;
    float* dv;
    float* dQKV;
    float* dqkv_bias;
    hipMalloc((void**)&dpadding_offset, sizeof(int) * batch_size * seq_len);
    hipMalloc((void**)&dhistory_length, sizeof(int) * batch_size);
    hipMalloc((void**)&dinput_length, sizeof(int) * batch_size);
    hipMalloc((void**)&dq, sizeof(float) * batch_size * seq_len * head_num * head_size);
    hipMalloc((void**)&dk, sizeof(float) * batch_size * seq_len * kv_head_num * head_size);
    hipMalloc((void**)&dv, sizeof(float) * batch_size * seq_len * kv_head_num * head_size);
    hipMalloc((void**)&dQKV, sizeof(float) * token_num * (head_num + 2 * kv_head_num) * head_size);
    hipMalloc((void**)&dqkv_bias, sizeof(float) * (head_num + 2 * kv_head_num) * head_size);

    hipMemcpy(dinput_length, input_length, sizeof(int) * batch_size, hipMemcpyHostToDevice);
    hipMemcpy(dhistory_length, history_length, sizeof(int) * batch_size, hipMemcpyHostToDevice);
    hipMemcpy(dpadding_offset, padding_offset, sizeof(int) * seq_len * batch_size, hipMemcpyHostToDevice);
    hipMemcpy(dQKV, QKV, sizeof(float) * token_num * (head_num + 2 * kv_head_num) * head_size, hipMemcpyHostToDevice);
    hipMemcpy(dqkv_bias, qkv_bias, sizeof(float) * (head_num + 2 * kv_head_num) * head_size, hipMemcpyHostToDevice);
    
    DataType type = getTensorType<float>(); 
    TensorWrapper<float>* q_buf = new TensorWrapper<float>(Device::GPU, type, {batch_size, head_num, seq_len, head_size}, dq);
    TensorWrapper<float>* k_buf = new TensorWrapper<float>(Device::GPU, type, {batch_size, kv_head_num, seq_len, head_size}, dk);
    TensorWrapper<float>* v_buf = new TensorWrapper<float>(Device::GPU, type, {batch_size, kv_head_num, seq_len, head_size}, dv);
    TensorWrapper<float>* QKV_buf = new TensorWrapper<float>(Device::GPU, type, {token_num, head_num + 2 * kv_head_num, head_size}, dQKV);
//    Tensor qkv_bias_buf(Device::GPU, type, {(head_num + 2 * kv_head_num), head_size}, dqkv_bias);
    LLaMAattentionWeights<float> attn_weights;
    attn_weights.qkv.bias = dqkv_bias;
    DataType type_int = getTensorType<int>(); 
    TensorWrapper<int>* input_length_buf = new TensorWrapper<int>(Device::GPU, type_int, {batch_size}, dinput_length);
    TensorWrapper<int>* history_length_buf = new TensorWrapper<int>(Device::GPU, type_int, {batch_size}, dhistory_length);
    TensorWrapper<int>* padding_offset_buf = new TensorWrapper<int>(Device::GPU, type_int, {batch_size, seq_len}, dpadding_offset);
    LLaMAAttentionStaticParams params;
    params.rotary_embedding_dim = rotary_embedding_dim;
    params.rotary_embedding_base = rotary_embedding_base;
    params.max_position_embeddings = max_position_embeddings;
    params.use_dynamic_ntk = false;
    // debug info, better to retain: 
    std::cout << "before launch kernel" << std::endl;
    launchAddFusedQKVBiasTransposeAndRoPE(q_buf,
                                          k_buf,
                                          v_buf,
                                          QKV_buf,
                                          attn_weights.qkv,
                                          padding_offset_buf,
                                          history_length_buf,
                                          input_length_buf,
                                          params);
    // debug info, better to retain: 
    std::cout << "after launch kernel" << std::endl;
    // debug info, better to retain: 
    std::cout << "cuda memcpy device to host" << std::endl;
    // Note: remember to memcpy from device to host and define the correct copy size(mul the sizeof(dtype)), or will cause segment fault
    CHECK(hipMemcpy(q, dq, sizeof(float) * batch_size * seq_len * head_num * head_size, hipMemcpyDeviceToHost));
    CHECK(hipMemcpy(k, dk, sizeof(float) * batch_size * seq_len * kv_head_num * head_size, hipMemcpyDeviceToHost));
    
    std::cout << "after memcpyd2h, dq[0] = " << q[0] << std::endl;
    std::cout << "before CPU function" << std::endl;
    float* hq = (float*)malloc(sizeof(float) * batch_size * seq_len * head_num * head_size); //output
    float* hk = (float*)malloc(sizeof(float) * batch_size * seq_len * kv_head_num * head_size); //output
    CPUfunc(hq,
            hk, //output
            v,
            QKV,
            qkv_bias,
            padding_offset,
            history_length,
            input_length,
            batch_size,
            seq_len,
            token_num,
            head_num,
            kv_head_num,
            head_size,
            rotary_embedding_dim,
            rotary_embedding_base);
    std::cout << "after CPU function" << std::endl;
    bool is_right = CheckResult(q, k, hq, hk, 
                                    batch_size * seq_len * head_num * head_size, 
                                            batch_size * seq_len * kv_head_num * head_size);
    // debug info, better to retain: 
    std::cout << "before free" << std::endl;
    std::cout << "passed" << std::endl;
    free(q);
    free(k);
    free(v);
    free(QKV);
    free(qkv_bias);
    free(padding_offset);
    free(history_length);
    free(input_length);
    free(hq);
    free(hk);
    hipFree(dq);
    hipFree(dk);
    hipFree(dv);
    hipFree(dQKV);
    hipFree(dqkv_bias);
    hipFree(dpadding_offset);
    hipFree(dhistory_length);
    hipFree(dinput_length);
}
