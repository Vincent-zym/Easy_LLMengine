#include "hip/hip_runtime.h"
#include <algorithm> // std::fill_n
#include <iostream>  // snprintf
#include <math.h>    // expf, log
#include <stdlib.h>  // rand
#include <string>    // std::string
#include <vector>    // std::vector

#include "src/kernels/fused_decoder_self_attention.h"
#include "src/utils/macro.h"

// bug1: MUST add CHECK to hipMemcpy to see if its work well
// (Vincent)note: this CPU implementation still exists bugs.
// when you are implementing LLMs inference on CPU, you can reuse the CPU kernel and test its correctness
// `./test_fused_decoder_attention` to test fp32 kernel
template <typename T>
void CPUMaskedAttn(T *q,
                   T *k,
                   T *v,
                   T *k_cache,
                   T *v_cache,
                   float *mha_output,
                   const int batch_size,
                   const int num_heads,
                   const int head_size,
                   int step)
{
    int batch_stride = num_heads * head_size;
    int head_stride = head_size;
    int cache_offset = batch_size * batch_stride;
    int block_nums = batch_size * num_heads;
    float scale = rsqrt(float(head_size));

    const T *q_mem = q;
    const T *k_mem = k;
    const T *v_mem = v;

    // tmp buffer
    float *sqk = (float *)malloc(sizeof(float) * (block_nums * (3 * head_size + step)));
    float *sq = sqk;
    float *sk = sq + block_nums * head_size;
    float *logits = sk + block_nums * head_size;
    float *sv = logits + block_nums * step;
    for (int batch_id = 0; batch_id < batch_size; batch_id++)
    {
        for (int head_id = 0; head_id < num_heads; head_id++)
        {
            float row_max = 0.0f;
            for (int iter = 0; iter < step; iter++)
            {
                float attn_score = 0.0f;
                for (int tid = 0; tid < head_size; tid++)
                {
                    int qkv_offset = batch_id * batch_stride + head_id * head_stride + tid;
                    // note: sq and sk's offset should be qkv_offset , not tid
                    sk[qkv_offset] = (float)k_cache[iter * cache_offset + qkv_offset];
                    // when final step, update k cache
                    if (iter == step - 1)
                    {
                        // TODO: update k cache with k with bias add
                        k_cache[iter * cache_offset + qkv_offset] = k_mem[qkv_offset];
                        sk[qkv_offset] = (float)k_mem[qkv_offset];
                    }

                    sq[qkv_offset] = (float)q_mem[qkv_offset];
                    float qk = sq[qkv_offset] * sk[qkv_offset] * scale;
                    // block reduce using multi warp reduce
                    // TODO: maybe broadcast the attn score to each thread of the block in blockreducesum
                    attn_score += qk;
                }
                // note: logtis's offset should be as follow, not should mul head size with iter
                // debug info,printf("every step/seqlen attn score = %f\n", attn_score);
                logits[batch_id * num_heads * step + head_id * step + iter] = attn_score;
                // softmax(logits), logits.shape = [bs, num heads, 1, step]
                row_max = std::max(attn_score, row_max);
            }
            //            printf("all step/seqlen(one row) max attn score = %f\n", row_max);
            float fenzi = 0.0f;
            float fenmu = 0.0f;
            for (int iter = 0; iter < step; iter++)
            { // row
                fenzi = expf(logits[batch_id * num_heads * step + head_id * step + iter] - row_max);
                fenmu += fenzi;
            }
            for (int iter = 0; iter < step; iter++)
            { // row
                logits[batch_id * num_heads * step + head_id * step + iter] = (float)(fenzi / fenmu);
                //              printf("logits=%f\n", fenzi / fenmu);
            }
            // logits*V = [bs, num heads, 1, step] * [mx_seq_len or step, bs, num heads, head size]
            // for(int iter = 0; iter < step; iter++) {
            for (int tid = 0; tid < head_size; tid++)
            {
                float O = 0.0f;
                int qkv_offset = batch_id * batch_stride + head_id * head_stride + tid;
                for (int iter = 0; iter < step; iter++)
                {
                    sv[qkv_offset] = (float)v_cache[iter * cache_offset + qkv_offset];
                    // when final step, update k cache
                    if (iter == step - 1)
                    {
                        // TODO: update k cache with k with bias add
                        v_cache[iter * cache_offset + qkv_offset] = v_mem[qkv_offset];
                        sv[qkv_offset] = (float)v_mem[qkv_offset];
                    }
                    O += sv[qkv_offset] * logits[batch_id * num_heads * step + head_id * step + iter];
                    //                printf("logits[%d]=%f, sv[%d]=%f, O=%f\n", iter, logits[iter], qkv_offset, sv[qkv_offset], O);
                }
                mha_output[qkv_offset] = O;
            }
        }
    }

    free(sqk);
}
template <typename T>
bool CheckResult(float *CPUoutput, T *GPUoutput, int output_size)
{
    for (int i = 0; i < output_size; i++)
    {
        float GPUres = (float)GPUoutput[i];
        if (fabs(CPUoutput[i] - GPUres) > 1e-6)
        {
            printf("the %dth res is wrong, CPUoutput = %f, GPUoutput = %f\n", i, CPUoutput[i], GPUres);
            return false;
        }
    }
    return true;
}
// (Vincent)note: when we develop fp16 fused decoder attention, we can enable the disabled macro to generalize fp16 an fp32 UT
// #define LAUNCH_FUSED_ATTN(dtype)                                                                                                      \
//     dtype *h_qkv;                                                                                                                     \
//     dtype *d_qkv;                                                                                                                     \
//     int qkv_size = batch_size * (2 * kv_num_heads + num_heads) * head_size;                                                           \
//     h_qkv = (dtype *)malloc(sizeof(dtype) * qkv_size);                                                                                \
//     hipMalloc((void **)&d_qkv, sizeof(dtype) * qkv_size);                                                                            \
//     dtype *h_kcache;                                                                                                                  \
//     dtype *d_kcache;                                                                                                                  \
//     int kcache_size = max_seq_len * batch_size * kv_num_heads * head_size;                                                            \
//     h_kcache = (dtype *)malloc(sizeof(dtype) * kcache_size);                                                                          \
//     hipMalloc((void **)&d_kcache, sizeof(dtype) * kcache_size);                                                                      \
//     dtype *h_vcache;                                                                                                                  \
//     dtype *d_vcache;                                                                                                                  \
//     int vcache_size = max_seq_len * batch_size * kv_num_heads * head_size;                                                            \
//     h_vcache = (dtype *)malloc(sizeof(dtype) * vcache_size);                                                                          \
//     hipMalloc((void **)&d_vcache, sizeof(dtype) * vcache_size);                                                                      \
//     for (int i = 0; i < qkv_size; i++)                                                                                                \
//     {                                                                                                                                 \
//         if (i < batch_size * num_heads * head_size)                                                                                   \
//         {                                                                                                                             \
//             if (i < batch_size * num_heads * head_size / 2)                                                                           \
//             {                                                                                                                         \
//                 h_qkv[i] = (dtype)(i + 1);                                                                                            \
//             }                                                                                                                         \
//             else                                                                                                                      \
//             {                                                                                                                         \
//                 h_qkv[i] = (dtype)(i + 1) / (dtype)10;                                                                                       \
//             }                                                                                                                         \
//         }                                                                                                                             \
//         else if (i < batch_size * (num_heads + kv_num_heads) * head_size)                                                             \
//         {                                                                                                                             \
//             if (i < batch_size * (num_heads + kv_num_heads / 2) * head_size)                                                          \
//             {                                                                                                                         \
//                 h_qkv[i] = (dtype)(i + 5);                                                                                            \
//             }                                                                                                                         \
//             else                                                                                                                      \
//             {                                                                                                                         \
//                 h_qkv[i] = (dtype)(i + 1) / (dtype)10;                                                                                       \
//             }                                                                                                                         \
//         }                                                                                                                             \
//         else if (i < batch_size * (num_heads + kv_num_heads * 2) * head_size)                                                         \
//         {                                                                                                                             \
//             if (i < batch_size * (num_heads + kv_num_heads + kv_num_heads / 2) * head_size)                                           \
//             {                                                                                                                         \
//                 h_qkv[i] = (dtype)(i - 3);                                                                                            \
//             }                                                                                                                         \
//             else                                                                                                                      \
//             {                                                                                                                         \
//                 h_qkv[i] = (dtype)(i - 7) / (dtype)10;                                                                                       \
//             }                                                                                                                         \
//         }                                                                                                                             \
//         printf("h_qkv[%d]= %f \n", i, (dtype)h_qkv[i]);                                                                                      \
//     }                                                                                                                                 \
//     dtype *h_q = h_qkv;                                                                                                               \
//     dtype *h_k = h_q + batch_size * num_heads * head_size;                                                                            \
//     dtype *h_v = h_k + batch_size * (kv_num_heads + num_heads) * head_size;                                                           \
//     for (int i = 0; i < (kcache_size * h_step) / max_seq_len; i++)                                                                    \
//     {                                                                                                                                 \
//         if (i < kcache_size / 2)                                                                                                      \
//         {                                                                                                                             \
//             h_kcache[i] = (dtype)(i + 1);                                                                                             \
//             h_vcache[i] = (dtype)(i + 1);                                                                                             \
//         }                                                                                                                             \
//         else                                                                                                                          \
//         {                                                                                                                             \
//             h_kcache[i] = (dtype)(i - kcache_size / 2 + 1) / (dtype)10;                                                                      \
//             h_vcache[i] = (dtype)(i - kcache_size / 2 + 1) / (dtype)10;                                                                      \
//         }                                                                                                                             \
//         printf("h_kcache[%d]= %f \n", i, (dtype)h_kcache[i]);                                                                                \
//         printf("h_vcache[%d]= %f \n", i, (dtype)h_vcache[i]);                                                                                \
//     }                                                                                                                                 \
//     dtype *h_o;                                                                                                                       \
//     dtype *d_o;                                                                                                                       \
//     int o_size = batch_size * num_heads * head_size;                                                                                  \
//     h_o = (dtype *)malloc(sizeof(dtype) * o_size);                                                                                    \
//     hipMalloc((void **)&d_o, sizeof(dtype) * o_size);                                                                                \
//     bool *h_finished = (bool *)malloc(sizeof(bool) * batch_size);                                                                     \
//     bool *d_finished;                                                                                                                 \
//     hipMalloc((void **)&d_finished, sizeof(bool) * batch_size);                                                                      \
//     for (int i = 0; i < batch_size; i++)                                                                                              \
//     {                                                                                                                                 \
//         h_finished[i] = static_cast<bool>(0);                                                                                         \
//     }                                                                                                                                 \
//     dtype *h_qkv_bias = (dtype *)malloc(sizeof(dtype) * (2 * kv_num_heads + num_heads) * head_size);                                  \
//     dtype *d_qkv_bias;                                                                                                                \
//     hipMalloc((void **)&d_qkv_bias, sizeof(dtype) * (2 * kv_num_heads + num_heads) * head_size);                                     \
//     for (int i = 0; i < (2 * kv_num_heads + num_heads) * head_size; i++)                                                              \
//     {                                                                                                                                 \
//         h_qkv_bias[i] = (dtype)0.0f;                                                                                                  \
//     }                                                                                                                                 \
//     hipMemcpy(d_qkv, h_qkv, sizeof(dtype) * batch_size * (2 * kv_num_heads + num_heads) * head_size, hipMemcpyHostToDevice);        \
//     hipMemcpy(d_qkv_bias, h_qkv_bias, sizeof(dtype) * (2 * kv_num_heads + num_heads) * head_size, hipMemcpyHostToDevice);           \
//     hipMemcpy(d_finished, h_finished, sizeof(bool) * batch_size, hipMemcpyHostToDevice);                                            \
//     hipMemcpy(d_kcache, h_kcache, sizeof(dtype) * kcache_size, hipMemcpyHostToDevice);                                              \
//     hipMemcpy(d_vcache, h_vcache, sizeof(dtype) * vcache_size, hipMemcpyHostToDevice);                                              \
//     DataType type = getTensorType<dtype>();                                                                                           \
//     DataType type_bool = getTensorType<bool>();                                                                                       \
//     DataType type_int = getTensorType<int>();                                                                                         \
//     TensorWrapper<dtype> *qkv = new TensorWrapper<dtype>(GPU, type, {batch_size, num_heads + 2 * kv_num_heads, head_size}, d_qkv);    \
//     TensorWrapper<dtype> *kcache = new TensorWrapper<dtype>(GPU, type, {h_layer_id, batch_size, kv_num_heads, max_seq_len, head_size}, d_kcache); \
//     TensorWrapper<dtype> *vcache = new TensorWrapper<dtype>(GPU, type, {h_layer_id, batch_size, kv_num_heads, max_seq_len, head_size}, d_vcache); \
//     TensorWrapper<bool> *finished = new TensorWrapper<bool>(GPU, type_bool, {batch_size}, d_finished);                                \
//     TensorWrapper<int> *step = new TensorWrapper<int>(CPU, type_int, {1}, &h_step);                                                   \
//     TensorWrapper<int> *layer_id = new TensorWrapper<int>(CPU, type_int, {1}, &h_layer_id);                                           \
//     TensorWrapper<dtype> *mha_output = new TensorWrapper<dtype>(GPU, type, {batch_size, num_heads, head_size}, d_o);                  \
//     BaseWeight<dtype> qkv_weight;                                                                                                     \
//     qkv_weight.bias = d_qkv_bias;                                                                                                     \
//     LLaMAAttentionStaticParams params;                                                                                                \
//     params.rotary_embedding_dim = rotary_embedding_dim;                                                                               \
//     params.rotary_embedding_base = rotary_embedding_base;                                                                             \
//     params.max_position_embeddings = max_position_embeddings;                                                                         \
//     params.use_dynamic_ntk = false;                                                                                                   \
//     launchDecoderMaskedMHA(qkv, qkv_weight, layer_id, kcache, vcache, finished, step, mha_output, params);                            \
//     CHECK(hipMemcpy(h_o, d_o, sizeof(dtype) * o_size, hipMemcpyDeviceToHost));                                                      \
//     float *CPU_output = (float *)malloc(sizeof(float) * o_size);                                                                      \
//     CPUMaskedAttn<dtype>(h_q, h_k, h_v, h_kcache, h_vcache, CPU_output, batch_size, num_heads, head_size, h_step);                    \
//     bool is_true = CheckResult<dtype>(CPU_output, h_o, o_size);                                                                       \
//     if (is_true)                                                                                                                      \
//     {                                                                                                                                 \
//         printf("test passed");                                                                                                        \
//     }                                                                                                                                 \
//     else                                                                                                                              \
//     {                                                                                                                                 \
//         printf("test failed");                                                                                                        \
//     }                                                                                                                                 \
//     free(h_qkv);                                                                                                                      \
//     free(h_kcache);                                                                                                                   \
//     free(h_vcache);                                                                                                                   \
//     free(h_o);                                                                                                                        \
//     free(CPU_output);                                                                                                                 \
//     free(h_finished);                                                                                                                 \
//     hipFree(d_finished);                                                                                                             \
//     hipFree(d_qkv);                                                                                                                  \
//     hipFree(d_o);                                                                                                                    \
//     hipFree(d_kcache);                                                                                                               \
//     hipFree(d_vcache);

int main(int argc, char *argv[])
{
    constexpr int batch_size = 1;
    constexpr int head_size = 4;
    constexpr int num_heads = 2;
    constexpr int kv_num_heads = 2;
    constexpr int max_seq_len = 4;
    int h_step = 4;
    int h_layer_id = 0;
    int rotary_embedding_dim = 128;
    float rotary_embedding_base = 10000;
    int max_position_embeddings = 2048;
    bool use_dynamic_ntk = false; // for dyn scaling rope
    float *h_qkv;                                                                                                                                 
    float *d_qkv;                                                                                                                                 
    int qkv_size = batch_size * (2 * kv_num_heads + num_heads) * head_size;                                                                       
    h_qkv = (float *)malloc(sizeof(float) * qkv_size);                                                                                            
    hipMalloc((void **)&d_qkv, sizeof(float) * qkv_size);                                                                                        
    float *h_kcache;                                                                                                                              
    float *d_kcache;                                                                                                                              
    int kcache_size = max_seq_len * batch_size * kv_num_heads * head_size;                                                                        
    h_kcache = (float *)malloc(sizeof(float) * kcache_size);                                                                                      
    hipMalloc((void **)&d_kcache, sizeof(float) * kcache_size);                                                                                  
    float *h_vcache;                                                                                                                              
    float *d_vcache;                                                                                                                              
    int vcache_size = max_seq_len * batch_size * kv_num_heads * head_size;                                                                        
    h_vcache = (float *)malloc(sizeof(float) * vcache_size);                                                                                      
    hipMalloc((void **)&d_vcache, sizeof(float) * vcache_size);                                                                                  
    for (int i = 0; i < qkv_size; i++)                                                                                                            
    {                                                                                                                                             
        if (i < batch_size * num_heads * head_size)                                                                                               
        {                                                                                                                                         
            if (i < batch_size * num_heads * head_size / 2)                                                                                       
            {                                                                                                                                     
                h_qkv[i] = (float)(i + 1);                                                                                                        
            }                                                                                                                                     
            else                                                                                                                                  
            {                                                                                                                                     
                h_qkv[i] = (float)(i - 3) / 10;                                                                                                   
            }                                                                                                                                     
        }                                                                                                                                         
        else if (i < batch_size * (num_heads + kv_num_heads) * head_size)                                                                         
        {                                                                                                                                         
            if (i < batch_size * (num_heads + kv_num_heads / 2) * head_size)                                                                      
            {                                                                                                                                     
                h_qkv[i] = (float)(i + 5);                                                                                                        
            }                                                                                                                                     
            else                                                                                                                                  
            {                                                                                                                                     
                h_qkv[i] = (float)(i + 1) / 10;                                                                                                   
            }                                                                                                                                     
        }                                                                                                                                         
        else if (i < batch_size * (num_heads + kv_num_heads * 2) * head_size)                                                                     
        {                                                                                                                                         
            if (i < batch_size * (num_heads + kv_num_heads + kv_num_heads / 2) * head_size)                                                       
            {                                                                                                                                     
                h_qkv[i] = (float)(i - 3);                                                                                                        
            }                                                                                                                                     
            else                                                                                                                                  
            {                                                                                                                                     
                h_qkv[i] = (float)(i - 7) / 10;                                                                                                   
            }                                                                                                                                     
        }                                                                                                                                         
        printf("h_qkv[%d]= %f \n", i, h_qkv[i]);                                                                                                  
    }                                                                                                                                             
    float *h_q = h_qkv;                                                                                                                           
    float *h_k = h_q + batch_size * num_heads * head_size;                                                                                        
    float *h_v = h_k + batch_size * (kv_num_heads + num_heads) * head_size;                                                                       
    for (int i = 0; i < (kcache_size * h_step) / max_seq_len; i++)                                                                                
    {                                                                                                                                             
        if (i < kcache_size / 2)                                                                                                                  
        {                                                                                                                                         
            h_kcache[i] = (float)(i + 1);                                                                                                         
            h_vcache[i] = (float)(i + 1);                                                                                                         
        }                                                                                                                                        
        else                                                                                                                                     
        {                                                                                                                                        
            h_kcache[i] = (float)(i - kcache_size / 2 + 1) / 10;                                                                                 
            h_vcache[i] = (float)(i - kcache_size / 2 + 1) / 10;                                                                                 
        }                                                                                                                                        
        printf("h_kcache[%d]= %f\n", i, h_kcache[i]);                                                                                           
        printf("h_vcache[%d]= %f\n", i, h_vcache[i]);                                                                                           
    }                                                                                                                                            
    float *h_o;                                                                                                                                  
    float *d_o;                                                                                                                                  
    int o_size = batch_size * num_heads * head_size;                                                                                             
    h_o = (float *)malloc(sizeof(float) * o_size);                                                                                               
    hipMalloc((void **)&d_o, sizeof(float) * o_size);                                                                                           
    bool *h_finished = (bool *)malloc(sizeof(bool) * batch_size);                                                                                
    bool *d_finished;                                                                                                                            
    hipMalloc((void **)&d_finished, sizeof(bool) * batch_size);                                                                                 
    for (int i = 0; i < batch_size; i++)                                                                                                         
    {                                                                                                                                            
        h_finished[i] = static_cast<bool>(0);                                                                                                    
    }                                                                                                                                            
    float *h_qkv_bias = (float *)malloc(sizeof(float) * (2 * kv_num_heads + num_heads) * head_size);                                             
    float *d_qkv_bias;                                                                                                                           
    hipMalloc((void **)&d_qkv_bias, sizeof(float) * (2 * kv_num_heads + num_heads) * head_size);                                                
    for (int i = 0; i < (2 * kv_num_heads + num_heads) * head_size; i++)                                                                         
    {                                                                                                                                            
        h_qkv_bias[i] = (float)0.0f;                                                                                                             
    }                                                                                                                                            
    hipMemcpy(d_qkv, h_qkv, sizeof(float) * batch_size * (2 * kv_num_heads + num_heads) * head_size, hipMemcpyHostToDevice);                   
    hipMemcpy(d_qkv_bias, h_qkv_bias, sizeof(float) * (2 * kv_num_heads + num_heads) * head_size, hipMemcpyHostToDevice);                      
    hipMemcpy(d_finished, h_finished, sizeof(bool) * batch_size, hipMemcpyHostToDevice);                                                       
    hipMemcpy(d_kcache, h_kcache, sizeof(float) * kcache_size, hipMemcpyHostToDevice);                                                         
    hipMemcpy(d_vcache, h_vcache, sizeof(float) * vcache_size, hipMemcpyHostToDevice);                                                         
    DataType type = getTensorType<float>();                                                                                                      
    DataType type_bool = getTensorType<bool>();                                                                                                  
    DataType type_int = getTensorType<int>();                                                                                                    
    TensorWrapper<float> *qkv = new TensorWrapper<float>(GPU, type, {batch_size, num_heads + 2 * kv_num_heads, head_size}, d_qkv);               
    TensorWrapper<float> *kcache = new TensorWrapper<float>(GPU, type, {h_layer_id, batch_size, kv_num_heads, max_seq_len, head_size}, d_kcache);
    TensorWrapper<float> *vcache = new TensorWrapper<float>(GPU, type, {h_layer_id, batch_size, kv_num_heads, max_seq_len, head_size}, d_vcache);
    TensorWrapper<bool> *finished = new TensorWrapper<bool>(GPU, type_bool, {batch_size}, d_finished);                                           
    TensorWrapper<int> *step = new TensorWrapper<int>(CPU, type_int, {1}, &h_step);                                                              
    TensorWrapper<int> *layer_id = new TensorWrapper<int>(CPU, type_int, {1}, &h_layer_id);                                                      
    TensorWrapper<float> *mha_output = new TensorWrapper<float>(GPU, type, {batch_size, num_heads, head_size}, d_o);                             
    BaseWeight<float> qkv_weight;                                                                                                                
    qkv_weight.bias = d_qkv_bias;                                                                                                                
    LLaMAAttentionStaticParams params;                                                                                                           
    params.rotary_embedding_dim = rotary_embedding_dim;                                                                                          
    params.rotary_embedding_base = rotary_embedding_base;                                                                                        
    params.max_position_embeddings = max_position_embeddings;                                                                                    
    params.use_dynamic_ntk = false;                                                                                                              
    launchDecoderMaskedMHA(qkv, qkv_weight, layer_id, kcache, vcache, finished, step, mha_output, params);                                       
    CHECK(hipMemcpy(h_o, d_o, sizeof(float) * o_size, hipMemcpyDeviceToHost));                                                                 
    float *CPU_output = (float *)malloc(sizeof(float) * o_size);                                                                                 
    CPUMaskedAttn<float>(h_q, h_k, h_v, h_kcache, h_vcache, CPU_output, batch_size, num_heads, head_size, h_step);                               
    bool is_true = CheckResult<float>(CPU_output, h_o, o_size);                                                                                  
    if (is_true)                                                                                                                                 
    {                                                                                                                                            
        printf("test passed\n");                                                                                                                   
    }                                                                                                                                            
    else                                                                                                                                         
    {                                                                                                                                            
        printf("test failed\n");                                                                                                                   
    }                                                                                                                                            
    free(h_qkv);                                                                                                                                 
    free(h_kcache);                                                                                                                              
    free(h_vcache);                                                                                                                              
    free(h_o);                                                                                                                                   
    free(CPU_output);                                                                                                                            
    free(h_finished);                                                                                                                            
    hipFree(d_finished);                                                                                                                        
    hipFree(d_qkv);                                                                                                                             
    hipFree(d_o);                                                                                                                               
    hipFree(d_kcache);                                                                                                                          
    hipFree(d_vcache);

}
