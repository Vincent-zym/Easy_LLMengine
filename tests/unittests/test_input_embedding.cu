#include "hip/hip_runtime.h"
#include <algorithm>   // std::fill_n
#include <iostream>    // snprintf
#include <math.h>      // expf, log
#include <stdlib.h>    // rand
#include <string>      // std::string
#include <vector>      // std::vector
#include <random>

#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>
#include <hip/hip_runtime.h>

#include "src/kernels/input_embedding.h"
// (Vincent)note:
// there is no embedding cpu kernel implementation now
// `./embedding` to test fp16 GPU kernel
// `./embedding 1` to test fp32 GPU kernel

#define CHECK(call)                                   \
do                                                    \
{                                                     \
    const hipError_t error_code = call;              \
    if (error_code != hipSuccess)                    \
    {                                                 \
        printf("CUDA Error:\n");                      \
        printf("    File:       %s\n", __FILE__);     \
        printf("    Line:       %d\n", __LINE__);     \
        printf("    Error code: %d\n", error_code);   \
        printf("    Error text: %s\n",                \
            hipGetErrorString(error_code));          \
        exit(1);                                      \
    }                                                 \
} while (0)

void cpuEmbedding(const int* input_ids, float* output, float* embed_table, const int max_context_token_num, const int hidden_size, const int vocab_size) {
    for (int i = 0; i < max_context_token_num; ++i) {
        for (int j = 0; j < hidden_size; ++j) {
            output[j + i * hidden_size] = embed_table[j + input_ids[i] * hidden_size];
        }
    }
}

bool checkResults(float* h_output, float* d_output, const int output_size) {
    float* d_output_cpu = (float*) malloc(output_size * sizeof(float)); // prepare for cpu check
    CHECK(hipMemcpy(d_output_cpu, d_output, output_size * sizeof(float), hipMemcpyDeviceToHost));
    for (int i = 0; i < output_size; ++i) {
        if (fabs(d_output_cpu[i] - h_output[i]) > 1e5) {
            std::cout << "Dev : ";
            for (int j = max(0, i - 10); j < min(output_size, i + 10); ++j) {
                std::cout << d_output_cpu[i];
            }
            std::cout << std::endl;
            std::cout << "Cpu : ";
            for (int j = max(0, i - 10); j < min(output_size, i + 10); ++j) {
                std::cout << h_output[i];
            }
            std::cout << std::endl;
            free(d_output_cpu);
            return false;
        }
    }
    free(d_output_cpu);
    return true;
}

int main(int argc, char *argv[]) {
    const int max_context_token_num = 64;
    const int hidden_size = 4096;
    const int vocab_size = 32000;
    const int input_size = max_context_token_num;
    const int table_size = vocab_size * hidden_size;
    const int output_size = max_context_token_num * hidden_size;

    int* h_input = (int*) malloc(input_size * sizeof(int));
    if (argv[1]) {
        float* h_table = (float*) malloc(table_size * sizeof(float));
        float* h_output = (float*) malloc(output_size * sizeof(float));

        // debug info, better to retain: 
        std::cout << "init memory on host" << std::endl;

        std::random_device rd;
        std::mt19937 gen(rd());
        std::uniform_int_distribution<> dis_int(0, vocab_size - 1);
        std::uniform_real_distribution<> dis_real(1.0, 2.0);

        for (int i = 0; i < max_context_token_num; ++i) {
            h_input[i] = dis_int(gen);
            printf("h_input[%d] = %d\n",i,  h_input[i]);
        }
        for (int i = 0; i < table_size; ++i) {
            h_table[i] = (float)(i / hidden_size);
        }

        int* d_input;
        float *d_table, *d_output;
        hipMalloc((void**)&d_input, input_size * sizeof(int));
        hipMalloc((void**)&d_table, table_size * sizeof(float));
        hipMalloc((void**)&d_output, output_size * sizeof(float));
        // debug info, better to retain: 
        std::cout << "init memory on device" << std::endl;

        CHECK(hipMemcpy(d_input, h_input, input_size * sizeof(int), hipMemcpyHostToDevice));
        CHECK(hipMemcpy(d_table, h_table, table_size * sizeof(float), hipMemcpyHostToDevice));
        // debug info, better to retain: 
        std::cout << "copy to device" << std::endl;

        DataType type_float = getTensorType<float>();
        DataType type_int = getTensorType<int>();
        TensorWrapper<int>* input_ids = new TensorWrapper<int>(Device::GPU, type_int, {max_context_token_num},    d_input);
        TensorWrapper<float>* output = new TensorWrapper<float>(Device::GPU, type_float, {max_context_token_num,     hidden_size}, d_output);
        EmbeddingWeight<float> emb_table;
        emb_table.data = d_table;
        launchInputEmbedding(input_ids, output, &emb_table);
        CHECK(hipMemcpy(h_output, output->data, output_size * sizeof(float), hipMemcpyDeviceToHost));
        std::cout << "printf h_output for check" << std::endl;
        for (int i = 0; i < max_context_token_num; i++){
            std::cout << (float)h_output[i * hidden_size] << std::endl;
        }

        hipFree(d_output);
        hipFree(d_table);
        hipFree(d_input);
        free(h_output);
        free(h_table);
        free(h_input);
    } else {
        half* h_table = (half*) malloc(table_size * sizeof(half));
        half* h_output = (half*) malloc(output_size * sizeof(half));

        // debug info, better to retain: 
        std::cout << "init memory on host" << std::endl;

        std::random_device rd;
        std::mt19937 gen(rd());
        std::uniform_int_distribution<> dis_int(0, vocab_size - 1);
        std::uniform_real_distribution<> dis_real(1.0, 2.0);

        for (int i = 0; i < max_context_token_num; ++i) {
            h_input[i] = dis_int(gen);
        }
	    printf("h_input[0] = %d\n", h_input[0]);
        for (int i = 0; i < table_size; ++i) {
            h_table[i] = (half)(i / hidden_size);
        }

        int* d_input;

        half *d_table, *d_output;
        hipMalloc((void**)&d_input, input_size * sizeof(int));
        hipMalloc((void**)&d_table, table_size * sizeof(half));
        hipMalloc((void**)&d_output, output_size * sizeof(half));
        // debug info, better to retain: 
        std::cout << "init memory on device" << std::endl;

        CHECK(hipMemcpy(d_input, h_input, input_size * sizeof(int), hipMemcpyHostToDevice));
        CHECK(hipMemcpy(d_table, h_table, table_size * sizeof(half), hipMemcpyHostToDevice));
        // debug info, better to retain: 
        std::cout << "copy to device" << std::endl;

        DataType type_float = getTensorType<float>();
        DataType type_half = getTensorType<half>();
        DataType type_int = getTensorType<int>();
        TensorWrapper<int>* input_ids = new TensorWrapper<int>(Device::GPU, type_int, {max_context_token_num},    d_input);
        TensorWrapper<half>* output = new TensorWrapper<half>(Device::GPU, type_half, {max_context_token_num,     hidden_size}, d_output);
        EmbeddingWeight<half> emb_table;
        emb_table.data = d_table;
        launchInputEmbedding(input_ids, output, &emb_table);
        CHECK(hipMemcpy(h_output, output->data, output_size * sizeof(half), hipMemcpyDeviceToHost));
        std::cout << "printf h_output for check" << std::endl;
        std::cout << (float)h_output[0] << std::endl;
        std::cout << (float)h_output[1] << std::endl;
        hipFree(d_output);
        hipFree(d_table);
        hipFree(d_input);
        free(h_output);
        free(h_table);
        free(h_input);        
    }
}
