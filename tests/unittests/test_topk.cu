#include "hip/hip_runtime.h"
#include <algorithm>   // std::fill_n
#include <iostream>    // snprintf
#include <math.h>      // expf, log
#include <stdlib.h>    // rand
#include <string>      // std::string
#include <vector>      // std::vector

#include <hip/hip_runtime.h>
#include "src/kernels/topK.h"
// (RussWong)note:
// there is no top k cpu kernel implementation now
// we compare the kernel correctnesss by eyes and result print infos
// `./test_topk` to test fp32 GPU kernel
int main() {
    const int batch_size = 1;
    const int vocab_size = 30000;
    const int beamwidth = 2;
    const int K = 5;
    const int BlockPerBeam = 8;
    // debug info, better to retain: std::cout <<"batch_size=" << batch_size << "  vocab_size=" << vocab_size << std::endl;
    const int probs_size = batch_size * vocab_size * beamwidth;
    float* h_probs;
    float *d_probs;
    h_probs = (float*)malloc(sizeof(float) * probs_size);
    hipMalloc((void**)&d_probs, sizeof(float) * probs_size);
    
    int topK_val_buf_size = batch_size * beamwidth * BlockPerBeam * K;
    int topK_ids_buf_size = batch_size * beamwidth * BlockPerBeam * K;
    int final_topK_val_buf_size = batch_size * beamwidth * K; // sampling topK buf size, beamsearch topK size = [batch_size * beam_width * beam_width]


    int *d_tmp_topk_ids;
    hipMalloc((void**)&d_tmp_topk_ids, sizeof(int) * topK_ids_buf_size);

    float *d_tmp_topk_vals;
    hipMalloc((void**)&d_tmp_topk_vals, sizeof(float) * topK_val_buf_size);

    int* h_final_topk_ids;
    int *d_final_topk_ids;
    h_final_topk_ids = (int*)malloc(sizeof(int) * final_topK_val_buf_size);
    hipMalloc((void**)&d_final_topk_ids, sizeof(int) * final_topK_val_buf_size);

    float* h_final_topk_vals;
    float *d_final_topk_vals;
    h_final_topk_vals = (float*)malloc(sizeof(float) * final_topK_val_buf_size);
    hipMalloc((void**)&d_final_topk_vals, sizeof(float) * final_topK_val_buf_size);

    for(int i = 0; i < probs_size; i++) { // 0-59999
       h_probs[i] = i;
    }
    hipMemcpy(d_probs, h_probs, sizeof(float)*probs_size, hipMemcpyHostToDevice);

    DataType type_float = getTensorType<float>();
    DataType type_int = getTensorType<int>();
    TensorWrapper<float>* probs_tensor = new TensorWrapper<float>(Device::GPU, 
                                                                type_float,
                                                                {batch_size * beamwidth, vocab_size}, 
                                                                d_probs);
    TensorWrapper<int> *tmp_topk_ids = new TensorWrapper<int>(Device::GPU, 
                                                                type_int,
                                                                {batch_size, beamwidth, BlockPerBeam, K}, 
                                                                d_tmp_topk_ids);
    TensorWrapper<float>* tmp_topk_vals = new TensorWrapper<float>(Device::GPU, 
                                                                type_float,
                                                                {batch_size, beamwidth, BlockPerBeam, K}, 
                                                                d_tmp_topk_vals);
    TensorWrapper<int> *final_topk_ids = new TensorWrapper<int>(Device::GPU, 
                                                                type_int,
                                                                {batch_size * beamwidth, K}, 
                                                                d_final_topk_ids);
    TensorWrapper<float> *final_topk_vals = new TensorWrapper<float>(Device::GPU, 
                                                                type_float,
                                                                {batch_size * beamwidth, K}, 
                                                                d_final_topk_vals);
    // debug info, better to retain: std::cout << "before launch kernel" << std::endl;
    launchTopKforBeamSearch(probs_tensor, tmp_topk_ids, tmp_topk_vals, final_topk_ids, final_topk_vals);
    // Note: remember to memcpy from device to host and define the correct copy size(mul the sizeof(dtype)), or will cause segment fault
    hipMemcpy(h_final_topk_ids, d_final_topk_ids, sizeof(int) * final_topK_val_buf_size, hipMemcpyDeviceToHost);
    hipMemcpy(h_final_topk_vals, d_final_topk_vals,  sizeof(float) * final_topK_val_buf_size, hipMemcpyDeviceToHost);
    for(int i = 0; i < final_topK_val_buf_size; i++) {
        int id = h_final_topk_ids[i];
        printf("topK id = %d\n", id);
        float val = h_final_topk_vals[i];
        printf("topK val =%f\n", val);
    }
    // debug info, better to retain: std::cout << "before free" << std::endl;
    free(h_probs);
    free(h_final_topk_ids);
    free(h_final_topk_vals);
    hipFree(d_probs);
    hipFree(d_final_topk_ids);
    hipFree(d_final_topk_vals);
    hipFree(d_tmp_topk_ids);
    hipFree(d_tmp_topk_vals);
}
